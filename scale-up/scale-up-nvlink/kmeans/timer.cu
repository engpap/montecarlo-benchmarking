#include "timer.h"

namespace kmeans {

timer::timer() {
    hipEventCreate(&m_start);
    hipEventCreate(&m_stop);
}

timer::~timer() {
    hipEventDestroy(m_start);
    hipEventDestroy(m_stop);
}

void timer::start() {
    hipEventRecord(m_start, 0);
}

float timer::stop() {
    float time;
    hipEventRecord(m_stop, 0);
    hipEventSynchronize(m_stop);
    hipEventElapsedTime(&time, m_start, m_stop);
    return time;
}

}

