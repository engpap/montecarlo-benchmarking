#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2012 University of A Coru�a
 *
 * CUSIMANN: An optimized simulated annealing software for GPUs
 *
 * Authors: A.M. Ferreiro, J.A. Garc�a, J.G. L�pez-Salas, C. V�zquez 
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/*
 * This sample minimizes the Griewank function (
 * http://www-optima.amp.i.kyoto-u.ac.jp/member/student/hedar/Hedar_files/TestGO_files/Page1905.htm ).
 * See supplied whitepaper for more explanations.
 */


#include "cusimann.cuh"
#include "nelderMead.h"

template <class T>
class Griewank {
public:
	__host__ __device__ T operator() (const T *x, unsigned int n, void *f_data) const
	{
		T f_x = 0.0f;
		T sum = 0.0f, prod = 1.0f;
	
		int i;
		for(i=0;i<n;i++)
			sum += pow(x[i],2)/4000.0f;
		
		for(i=0;i<n;i++)
			prod *= cos(x[i]/sqrt(i+1.0f));
	
		f_x = sum - prod + 1.0f;
	
		return f_x;
	}
};

double f_nelderMead(unsigned int n, const double *x, double *grad, void *f_data){
	return Griewank<double>()(x,n,f_data);
}

int main() {
	real T_0 = 1000000, T_min = 0.01;
	const unsigned int n = 3, N = 500;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*)malloc(sizeFD), f_cusimann_minimum;
	lb = (real*)malloc(sizeFD);
	unsigned int i;
	for(i=0;i<n;i++)
		lb[i] = -600; 
	ub = (real*)malloc(sizeFD);
	for(i=0;i<n;i++)
		ub[i] = 600;

	unsigned int n_threads_per_block = 256;
	unsigned int n_blocks = 64;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb, ub, Griewank<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double *nelderMead_minimum = (double*)malloc(n*sizeof(double)), f_nelderMead_minimum;
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);
	
	return EXIT_SUCCESS;
}
