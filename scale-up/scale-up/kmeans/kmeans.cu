#include "kmeans.h"
#include "util.h"
#include <thrust/reduce.h>

namespace kmeans {

int kmeans(int iterations,
           int n, int d, int k,
           thrust::device_vector<double>** data,
           thrust::device_vector<int>** labels,
           thrust::device_vector<double>** centroids,
           thrust::device_vector<double>** distances,
           int n_gpu,
           bool init_from_labels, 
           double threshold) {
    thrust::device_vector<double> *data_dots[16];
    thrust::device_vector<double> *centroid_dots[16];
    thrust::device_vector<double> *pairwise_distances[16];
    thrust::device_vector<int> *labels_copy[16];
    thrust::device_vector<int> *range[16];
    thrust::device_vector<int> *indices[16];
    thrust::device_vector<int> *counts[16];
    
    thrust::host_vector<double> h_centroids( k * d );
    thrust::host_vector<double> h_centroids_tmp( k * d );
    int h_changes[16], *d_changes[16];
    double h_distance_sum[16], *d_distance_sum[16];


    for (int q = 0; q < n_gpu; q++) {

        hipSetDevice(q);
        hipMalloc(&d_changes[q], sizeof(int));
        hipMalloc(&d_distance_sum[q], sizeof(double));
        detail::labels_init();
        data_dots[q] = new thrust::device_vector <double>(n/n_gpu);
        centroid_dots[q] = new thrust::device_vector<double>(n/n_gpu);
        pairwise_distances[q] = new thrust::device_vector<double>(n/n_gpu * k);
        labels_copy[q] = new thrust::device_vector<int>(n/n_gpu * d);
        range[q] = new thrust::device_vector<int>(n/n_gpu);
        counts[q] = new thrust::device_vector<int>(k);
        indices[q] = new thrust::device_vector<int>(n/n_gpu);
        //Create and save "range" for initializing labels
        thrust::copy(thrust::counting_iterator<int>(0),
                     thrust::counting_iterator<int>(n/n_gpu), 
                     (*range[q]).begin());

        detail::make_self_dots(n/n_gpu, d, *data[q], *data_dots[q]);
        if (init_from_labels) {
            detail::find_centroids(n/n_gpu, d, k, *data[q], *labels[q], *centroids[q], *range[q], *indices[q], *counts[q]);
        }
    }

    double prior_distance_sum = 0;
    int i=0;
    for(; i < iterations; i++) {
        //Average the centroids from each device
        if (n_gpu > 1) {
            for (int p = 0; p < k * d; p++) h_centroids[p] = 0.0;
            for (int q = 0; q < n_gpu; q++) {
                hipSetDevice(q);
                detail::memcpy(h_centroids_tmp, *centroids[q]);
                detail::streamsync(q);
                for (int p = 0; p < k * d; p++) h_centroids[p] += h_centroids_tmp[p];
            }
            for (int p = 0; p < k * d; p++) h_centroids[p] /= n_gpu;
            //Copy the averaged centroids to each device 
            for (int q = 0; q < n_gpu; q++) {
                hipSetDevice(q);
                detail::memcpy(*centroids[q],h_centroids);
            }
        }
        for (int q = 0; q < n_gpu; q++) {
            //TODO compute total distance
            hipSetDevice(q);
          
            detail::calculate_distances(n/n_gpu, d, k,
                                        *data[q], *centroids[q], *data_dots[q],
                                        *centroid_dots[q], *pairwise_distances[q]);

            detail::relabel(n/n_gpu, k, *pairwise_distances[q], *labels[q], *distances[q], d_changes[q]);
            //TODO remove one memcpy
            detail::memcpy(*labels_copy[q], *labels[q]);
            detail::find_centroids(n/n_gpu, d, k, *data[q], *labels[q], *centroids[q], *range[q], *indices[q], *counts[q]);
            detail::memcpy(*labels[q], *labels_copy[q]);
            //double d_distance_sum[q] = thrust::reduce(distances[q].begin(), distances[q].end())
            mycub::sum_reduce(*distances[q], d_distance_sum[q]);
        }
#if __VERBOSE
        double distance_sum = 0.0;
        for (int q = 0; q < n_gpu; q++) {
            hipMemcpyAsync(h_changes+q, d_changes[q], sizeof(int), hipMemcpyDeviceToHost, hip_stream[q]);
            hipMemcpyAsync(h_distance_sum+q, d_distance_sum[q], sizeof(double), hipMemcpyDeviceToHost, hip_stream[q]);
            detail::streamsync(q);
            std::cout << "Device " << q << ":  Iteration " << i << " produced " << h_changes[q]
                      << " changes and the total_distance is " << h_distance_sum[q] << std::endl;
            distance_sum += h_distance_sum[q];
        }
        if (i > 0) {
            double delta = distance_sum / prior_distance_sum;
            if (delta > 1 - threshold) {
                std::cout << "Threshold triggered. Terminating iterations early." << std::endl;
                return i + 1;
            }
        }
        prior_distance_sum = distance_sum;
#endif
        
    }
    for (int q = 0; q < n_gpu; q++) {
       hipSetDevice(q);
       hipFree(d_changes[q]);
       detail::labels_close();
       delete(pairwise_distances[q]);
       delete(data_dots[q]);
       delete(centroid_dots[q]);
    }
    return i;


}


}
