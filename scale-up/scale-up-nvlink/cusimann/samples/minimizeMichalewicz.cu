#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2012 University of A Coru�a
 *
 * CUSIMANN: An optimized simulated annealing software for GPUs
 *
 * Authors: A.M. Ferreiro, J.A. Garc�a, J.G. L�pez-Salas, C. V�zquez 
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/*
 * This sample minimizes the Michalewics Function (
 * http://www-optima.amp.i.kyoto-u.ac.jp/member/student/hedar/Hedar_files/TestGO_files/Page2376.htm ).
 * See supplied whitepaper for more explanations.
 */


#include "cusimann.cuh"
#include "nelderMead.h"

#define M_PI_CUDA 3.14159265358979323846f  /* pi */

template <class T>
class Michalewicz {
public:
	__host__ __device__ T operator() (const T *x, unsigned int n, void *f_data) const
	{
		T f_x = 0.0f, m=10.0f;
		
		int i;
		for(i=1;i<=n;i++)
			f_x += sin(x[i-1]) * pow(sin(i*pow(x[i-1],2)/M_PI_CUDA),2.0f*m);
		
		return -f_x;
	}
};

double f_nelderMead(unsigned int n, const double *x, double *grad, void *f_data){
	return Michalewicz<double>()(x,n,f_data);
}

int main() {
	real T_0 = 1000, T_min = 0.01;
	const unsigned int n = 10, N = 100;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*)malloc(sizeFD), f_cusimann_minimum;
	lb = (real*)malloc(sizeFD);
	unsigned int i;
	for(i=0;i<n;i++)
		lb[i] = 0; 
	ub = (real*)malloc(sizeFD);
	for(i=0;i<n;i++)
		ub[i] = M_PI;

	unsigned int n_threads_per_block = 256;
	unsigned int n_blocks = 64;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb, ub, Michalewicz<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double *nelderMead_minimum = (double*)malloc(n*sizeof(double)), f_nelderMead_minimum;
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);
	
	return EXIT_SUCCESS;
}
