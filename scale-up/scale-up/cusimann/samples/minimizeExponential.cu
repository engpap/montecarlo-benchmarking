#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2012 University of A Coru�a
 *
 * CUSIMANN: An optimized simulated annealing software for GPUs
 *
 * Authors: A.M. Ferreiro, J.A. Garc�a, J.G. L�pez-Salas, C. V�zquez 
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/*
 * This sample minimizes the Exponential function ( See L. Breiman, A. Cutler.:
 * A deterministic algorithm for global optimization. Math. Progr. 58, 179-199
 * (1993) ).
 * See supplied whitepaper for more explanations.
 */


#include "cusimann.cuh"
#include "nelderMead.h"

template <class T>
class Exponential {
public:
	__host__ __device__ T operator() (const T *x, unsigned int n, void *f_data) const
	{
		T f_x, s1 = 0.0f;

		int i;
		for(i=0;i<n;i++)
			s1 += pow(x[i],2);
	
		f_x = -exp(-0.5f*s1);

		return f_x;
	}
};

double f_nelderMead(unsigned int n, const double *x, double *grad, void *f_data){
	return Exponential<double>()(x,n,f_data);
}

int main() {
	real T_0 = 1000, T_min = 0.1;
	const unsigned int n = 2, N = 100;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*)malloc(sizeFD), f_cusimann_minimum;
	lb = (real*)malloc(sizeFD);
	unsigned int i;
	for(i=0;i<n;i++)
		lb[i] = -1; 
	ub = (real*)malloc(sizeFD);
	for(i=0;i<n;i++)
		ub[i] = 1;

	unsigned int n_threads_per_block = 256;
	unsigned int n_blocks = 64;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb, ub, Exponential<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double f_nelderMead_minimum;
	double *nelderMead_minimum = (double*)malloc(n*sizeof(double));
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);
	
	return EXIT_SUCCESS;
}
