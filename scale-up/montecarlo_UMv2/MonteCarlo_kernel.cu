#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

////////////////////////////////////////////////////////////////////////////////
// Global types
////////////////////////////////////////////////////////////////////////////////
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo_common.h"

////////////////////////////////////////////////////////////////////////////////
// Helper reduction template
// Please see the "reduction" CUDA Sample for more information
////////////////////////////////////////////////////////////////////////////////
#include "MonteCarlo_reduction.cuh"

////////////////////////////////////////////////////////////////////////////////
// Internal GPU-side data structures
////////////////////////////////////////////////////////////////////////////////
#define MAX_OPTIONS (1024 * 1024*32)

// Preprocessed input option data
typedef struct
{
    real S;
    real X;
    real MuByT;
    real VBySqrtT;
} __TOptionData;

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut payoff functions for different precision modes
////////////////////////////////////////////////////////////////////////////////
__device__ inline float endCallValue(float S, float X, float r, float MuByT, float VBySqrtT)
{
    float callValue = S * __expf(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0F) ? callValue : 0.0F;
}

__device__ inline double endCallValue(double S, double X, double r, double MuByT, double VBySqrtT)
{
    double callValue = S * exp(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0) ? callValue : 0.0;
}

#define THREAD_N 256

////////////////////////////////////////////////////////////////////////////////
// This kernel computes the integral over all paths using a single thread block
// per option. It is fastest when the number of thread blocks times the work per
// block is high enough to keep the GPU busy.
////////////////////////////////////////////////////////////////////////////////
static __global__ void MonteCarloOneBlockPerOption(
    hiprandState *__restrict rngStates,
    const __TOptionData *__restrict optionData,
    __TOptionValue *__restrict callValue,
    int pathN,
    int optionN)
{
    const int SUM_N = THREAD_N;
    __shared__ real s_SumCall[SUM_N];
    __shared__ real s_Sum2Call[SUM_N];

    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Copy random number state to local memory for efficiency
    hiprandState localState = rngStates[tid];
    for (int optionIndex = blockIdx.x; optionIndex < optionN; optionIndex += gridDim.x)
    {
        const real S = optionData[optionIndex].S;
        const real X = optionData[optionIndex].X;
        const real MuByT = optionData[optionIndex].MuByT;
        const real VBySqrtT = optionData[optionIndex].VBySqrtT;

        // Cycle through the entire samples array:
        // derive end stock price for each path
        // accumulate partial integrals into intermediate shared memory buffer
        for (int iSum = threadIdx.x; iSum < SUM_N; iSum += blockDim.x)
        {
            __TOptionValue sumCall = {0, 0};

#pragma unroll 8
            for (int i = iSum; i < pathN; i += SUM_N)
            {
                real r = hiprand_normal(&localState);
                real callValue = endCallValue(S, X, r, MuByT, VBySqrtT);
                sumCall.Expected += callValue;
                sumCall.Confidence += callValue * callValue;
            }

            s_SumCall[iSum] = sumCall.Expected;
            s_Sum2Call[iSum] = sumCall.Confidence;
        }

        // Reduce shared memory accumulators
        // and write final result to global memory
        sumReduce<real, SUM_N, THREAD_N>(s_SumCall, s_Sum2Call);

        if (threadIdx.x == 0)
        {
            __TOptionValue t = {s_SumCall[0], s_Sum2Call[0]};
            callValue[optionIndex] = t;
        }
    }
}

static __global__ void rngSetupStates(
    hiprandState *rngState,
    int device_id)
{
    // determine global thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each threadblock gets different seed,
    // Threads within a threadblock get different sequence numbers
    hiprand_init(blockIdx.x + gridDim.x * device_id, threadIdx.x, 0, &rngState[tid]);
}

////////////////////////////////////////////////////////////////////////////////
// Host-side interface to GPU Monte Carlo
////////////////////////////////////////////////////////////////////////////////

extern "C" void initMonteCarloGPU(TOptionPlan *plan)
{
    checkCudaErrors(hipMallocManaged(&plan->um_OptionData, sizeof(__TOptionData) * (plan->optionCount)));
    checkCudaErrors(hipMallocManaged(&plan->um_CallValue, sizeof(__TOptionValue) * (plan->optionCount)));

    // Applications can use hipMemAdviseSetAccessedBy performance hint with hipCpuDeviceId to enable direct access of GPU memory on supported systems.
    //checkCudaErrors(hipMemAdvise(plan->um_CallValue, sizeof(__TOptionValue) * (plan->optionCount), hipMemAdviseSetAccessedBy, hipCpuDeviceId)); 
    
    // 4120 CPU page faults
    // checkCudaErrors(hipMemAdvise(plan->um_OptionData, sizeof(__TOptionData) * (plan->optionCount), hipMemAdviseSetReadMostly, plan->device)); 
    

    
    // Allocate internal device memory
    // Allocate states for pseudo random number generators
    checkCudaErrors(hipMallocManaged((void **)&plan->rngStates,
                                      plan->gridSize * THREAD_N * sizeof(hiprandState)));

    // THIS WORKS!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! REMOVE COMMENT!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    hipMemAdvise(plan->rngStates, plan->gridSize * THREAD_N * sizeof(hiprandState), hipMemAdviseSetPreferredLocation, plan->device);

    // place each device pathN random numbers apart on the random number sequence
    rngSetupStates<<<plan->gridSize, THREAD_N>>>(plan->rngStates, plan->device);
    getLastCudaError("rngSetupStates kernel failed.\n");
}

// Compute statistics and deallocate internal device memory
extern "C" void closeMonteCarloGPU(TOptionPlan *plan)
{

    checkCudaErrors(hipMemAdvise(plan->um_CallValue, sizeof(__TOptionValue) * (plan->optionCount), hipMemAdviseSetPreferredLocation, hipCpuDeviceId));
    
    for (int i = 0; i < plan->optionCount; i++)
    {
        const double RT = plan->optionData[i].R * plan->optionData[i].T;
        const double sum = plan->um_CallValue[i].Expected;
        const double sum2 = plan->um_CallValue[i].Confidence;
        const double pathN = plan->pathN;
        // Derive average from the total sum and discount by riskfree rate
        plan->callValue[i].Expected = (float)(exp(-RT) * sum / pathN);
        // Standard deviation
        double stdDev = sqrt((pathN * sum2 - sum * sum) / (pathN * (pathN - 1)));
        // Confidence width; in 95% of all cases theoretical value lies within these borders
        plan->callValue[i].Confidence = (float)(exp(-RT) * 1.96 * stdDev / sqrt(pathN));
    }

    checkCudaErrors(hipFree(plan->rngStates));
    checkCudaErrors(hipFree(plan->um_CallValue));
    checkCudaErrors(hipFree(plan->um_OptionData));
}

// Main computations
extern "C" void MonteCarloGPU(TOptionPlan *plan, hipStream_t stream)
{   

    if (plan->optionCount <= 0 || plan->optionCount > MAX_OPTIONS)
    {
        printf("MonteCarloGPU(): bad option count.\n");
        return;
    }

    checkCudaErrors(hipMemAdvise(plan->um_OptionData, sizeof(__TOptionData) * (plan->optionCount), hipMemAdviseSetPreferredLocation, hipCpuDeviceId)); 

    __TOptionData *optionData = (__TOptionData *)plan->um_OptionData;

    for (int i = 0; i < plan->optionCount; i++)
    {
        const double T = plan->optionData[i].T;
        const double R = plan->optionData[i].R;
        const double V = plan->optionData[i].V;
        const double MuByT = (R - 0.5 * V * V) * T;
        const double VBySqrtT = V * sqrt(T);
        optionData[i].S = (real)plan->optionData[i].S;
        optionData[i].X = (real)plan->optionData[i].X;
        optionData[i].MuByT = (real)MuByT;
        optionData[i].VBySqrtT = (real)VBySqrtT;
    }

    checkCudaErrors(hipMemAdvise(plan->um_OptionData, sizeof(__TOptionData) * (plan->optionCount), hipMemAdviseSetAccessedBy, plan->device)); 

    // 1
    checkCudaErrors(hipMemAdvise(plan->um_CallValue, sizeof(__TOptionValue) * (plan->optionCount), hipMemAdviseSetPreferredLocation, plan->device));

    // 2
    //checkCudaErrors(hipMemAdvise(plan->um_CallValue, sizeof(__TOptionValue) * (plan->optionCount), hipMemAdviseSetPreferredLocation, hipCpuDeviceId));

    // 2
    //checkCudaErrors(hipMemAdvise(plan->um_CallValue, sizeof(__TOptionValue) * (plan->optionCount), hipMemAdviseSetAccessedBy, plan->device));

    MonteCarloOneBlockPerOption<<<plan->gridSize, THREAD_N, 0, stream>>>(
        plan->rngStates,
        (__TOptionData *)(plan->um_OptionData),
        (__TOptionValue *)(plan->um_CallValue),
        plan->pathN,
        plan->optionCount);
    getLastCudaError("MonteCarloOneBlockPerOption() execution failed\n");

}
