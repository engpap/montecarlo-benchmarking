#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/iterator/counting_iterator.h>

#include "labels.h"

__device__ double atomicAdd_dd(double* address, double val)
{
    unsigned long long int* address_as_ull =
        (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                    __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

namespace kmeans {
namespace detail {

__device__ __forceinline__ void update_centroid(int label, int dimension,
                                                int d,
                                                double accumulator, double* centroids,
                                                int count, int* counts) {
    int index = label * d + dimension;
    double* target = centroids + index;
    atomicAdd(target, accumulator);
    if (dimension == 0) {
        atomicAdd(counts + label, count);
    }             
}

__global__ void calculate_centroids(int n, int d, int k,
                                    double* data,
                                    int* ordered_labels,
                                    int* ordered_indices,
                                    double* centroids,
                                    int* counts) {
    int in_flight = blockDim.y * gridDim.y;
    int labels_per_row = (n - 1) / in_flight + 1; 
    for(int dimension = threadIdx.x; dimension < d; dimension += blockDim.x) {
        double accumulator = 0;
        int count = 0;
        int global_id = threadIdx.y + blockIdx.y * blockDim.y;
        int start = global_id * labels_per_row;
        int end = (global_id + 1) * labels_per_row;
        end = (end > n) ? n : end;
        int prior_label;
        if (start < n) {
            prior_label = ordered_labels[start];
        
            for(int label_number = start; label_number < end; label_number++) {
                int label = ordered_labels[label_number];
                if (label != prior_label) {
                    update_centroid(prior_label, dimension,
                                    d,
                                    accumulator, centroids,
                                    count, counts);
                    accumulator = 0;
                    count = 0;
                }
  
                double value = data[dimension + ordered_indices[label_number] * d];
                accumulator += value;
                prior_label = label;
                count++;
            }
            update_centroid(prior_label, dimension,
                            d,
                            accumulator, centroids,
                            count, counts);
        }
    }
}

__global__ void scale_centroids(int d, int k, int* counts, double* centroids) {
    int global_id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_id_y = threadIdx.y + blockIdx.y * blockDim.y;
    if ((global_id_x < d) && (global_id_y < k)) {
        int count = counts[global_id_y];
        //To avoid introducing divide by zero errors
        //If a centroid has no weight, we'll do no normalization
        //This will keep its coordinates defined.
        if (count < 1) {
            count = 1;
        }
        double scale = 1.0/double(count);
        centroids[global_id_x + d * global_id_y] *= scale;
    }
}

void find_centroids(int n, int d, int k,
                    thrust::device_vector<double>& data,
                    thrust::device_vector<int>& labels,
                    thrust::device_vector<double>& centroids,
                    thrust::device_vector<int>& range,
                    thrust::device_vector<int>& indices,
                    thrust::device_vector<int>& counts) {
    int dev_num;
    hipGetDevice(&dev_num);
    detail::memcpy(indices,range);
    //Bring all labels with the same value together
#if 0
    thrust::sort_by_key(labels.begin(),
                        labels.end(),
                        indices.begin());
#else
    mycub::sort_by_key_int(labels, indices);
#endif

    //Initialize centroids to all zeros
    detail::memzero(centroids);

    //Initialize counts to all zeros
    detail::memzero(counts);
    
    //Calculate centroids 
    int n_threads_x = 64;
    int n_threads_y = 16;
    //XXX Number of blocks here is hard coded at 30
    //This should be taken care of more thoughtfully.
    detail::calculate_centroids<<<dim3(1, 30), dim3(n_threads_x, n_threads_y),
                                  0, hip_stream[dev_num]>>>
        (n, d, k,
         thrust::raw_pointer_cast(data.data()),
         thrust::raw_pointer_cast(labels.data()),
         thrust::raw_pointer_cast(indices.data()),
         thrust::raw_pointer_cast(centroids.data()),
         thrust::raw_pointer_cast(counts.data()));
    
    //Scale centroids
    detail::scale_centroids<<<dim3((d-1)/32+1, (k-1)/32+1), dim3(32, 32),
                              0, hip_stream[dev_num]>>>
        (d, k,
         thrust::raw_pointer_cast(counts.data()),
         thrust::raw_pointer_cast(centroids.data()));
}

}
}
