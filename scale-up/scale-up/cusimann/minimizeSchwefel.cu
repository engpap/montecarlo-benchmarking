#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2012 University of A Coru�a
 *
 * CUSIMANN: An optimized simulated annealing software for GPUs
 *
 * Authors: A.M. Ferreiro, J.A. Garc�a, J.G. L�pez-Salas, C. V�zquez 
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/*
 * This sample minimizes the Normalized Schwefel function (
 * http://www.it.lut.fi/ip/evo/functions/node10.html ).
 * See supplied whitepaper for more explanations.
 */


#include "cusimann.cuh"
#include "nelderMead.h"

template <class T>
class Schwefel {
public:
	__host__ __device__ T operator() (const T *x, unsigned int n, void *f_data) const
	{
		T f_x = 0.0f;
	
		int i;
		for(i=0;i<n;i++)
			f_x = f_x + x[i] * sin(sqrt(fabs(x[i])));

		f_x = f_x / n;
	
		return - f_x;
	}
};

double f_nelderMead(unsigned int n, const double *x, double *grad, void *f_data){
	return Schwefel<double>()(x,n,f_data);
}

int main(int argc, char* argv[]) {
    if (argc != 2) 
    {
        printf("Specify the dimension of the searching space\n");
        exit(1);
    }
    const unsigned n = atoi(argv[1]);

    //real T_0 = 1000, T_min = 0.1;
    //const unsigned int n = 13, N = 500;

	real T_0 = 10000, T_min = 0.1;
	const unsigned int N = 500;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*)malloc(sizeFD), f_cusimann_minimum;
	lb = (real*)malloc(sizeFD);
	unsigned int i;
	for(i=0;i<n;i++)
		lb[i] = -512; 
	ub = (real*)malloc(sizeFD);
	for(i=0;i<n;i++)
		ub[i] = 512;

	unsigned int n_threads_per_block = 256;
    //unsigned int n_blocks = 64;
	unsigned int n_blocks = 256;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb, ub, Schwefel<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double *nelderMead_minimum = (double*)malloc(n*sizeof(double)), f_nelderMead_minimum;
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);

	return EXIT_SUCCESS;
}
