#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2012 University of A Coru�a
 *
 * CUSIMANN: An optimized simulated annealing software for GPUs
 *
 * Authors: A.M. Ferreiro, J.A. Garc�a, J.G. L�pez-Salas, C. V�zquez 
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/*
 * This sample minimizes the Shubert Function (
 * http://www-optima.amp.i.kyoto-u.ac.jp/member/student/hedar/Hedar_files/TestGO_files/Page1882.htm ).
 * See supplied whitepaper for more explanations.
 */


#include "cusimann.cuh"
#include "nelderMead.h"

template <class T>
class Shubert {
public:
	__host__ __device__ T operator() (const T *x, unsigned int n, void *f_data) const
	{
		T f_x, sum, prod = 1.0f;
	
		int i,j;
		for(i=0;i<n;i++){
			sum = 0.0f;
			for(j=1;j<=5;j++)
				sum += j*cos((j+1)*x[i]+j);
			prod *= sum;
		}

		f_x = prod;

		return f_x;
	}
};

double f_nelderMead(unsigned int n, const double *x, double *grad, void *f_data){
	return Shubert<double>()(x,n,f_data);
}

int main() {
	float T_0 = 1000, T_min = 0.1;
	const unsigned int n = 2, N = 100;
	const float rho = 0.99;
	size_t sizeFD = n * sizeof(float);
	float *lb, *ub, *cusimann_minimum = (float*)malloc(sizeFD), f_cusimann_minimum;
	lb = (float*)malloc(sizeFD);
	unsigned int i;
	for(i=0;i<n;i++)
		lb[i] = -10; 
	ub = (float*)malloc(sizeFD);
	for(i=0;i<n;i++)
		ub[i] = 10;

	unsigned int n_threads_per_block = 256;
	unsigned int n_blocks = 64;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb, ub, Shubert<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double *nelderMead_minimum = (double*)malloc(n*sizeof(double)), f_nelderMead_minimum;
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);
	
	return EXIT_SUCCESS;
}
