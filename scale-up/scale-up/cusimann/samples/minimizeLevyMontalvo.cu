#include "hip/hip_runtime.h"
/* Copyright (c) 2011-2012 University of A Coru�a
 *
 * CUSIMANN: An optimized simulated annealing software for GPUs
 *
 * Authors: A.M. Ferreiro, J.A. Garc�a, J.G. L�pez-Salas, C. V�zquez 
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the
 * "Software"), to deal in the Software without restriction, including
 * without limitation the rights to use, copy, modify, merge, publish,
 * distribute, sublicense, and/or sell copies of the Software, and to
 * permit persons to whom the Software is furnished to do so, subject to
 * the following conditions:
 *
 * The above copyright notice and this permission notice shall be
 * included in all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
 * NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
 * OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
 * WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/*
 * This sample minimizes the Levy and Montalvo function ( See A.V. Levy, A. Montalvo.:
 * The tunneling algorithm for the global minimization of functions. SIAM J. Sci and
 * Stat. Comput. 6, 15-29 (1985) ).
 * See supplied whitepaper for more explanations.
 */


#include "cusimann.cuh"
#include "nelderMead.h"

#define M_PI_CUDA 3.14159265358979323846f  /* pi */

template <class T>
class LevyMontalvo {
public:
	__host__ __device__ T operator() (const T *x, unsigned int n, void *f_data) const
	{
		T f_x = 0.0f, aux1, aux2 = 0.0f;
	
		T y0, ynMinus1, yi, yiPlus1;

		y0 = 1.0f + (x[0]+1.0f)/4.0f;
		aux1 = 10.0f*pow(sin(M_PI_CUDA*y0),2);
	
		int i;
		for(i=0;i<n-1;i++){
			yi = 1.0f + (x[i]+1.0f)/4.0f;
			yiPlus1 = 1.0f + (x[i+1]+1.0f)/4.0f;
			aux2 += pow(yi-1.0f,2)*(1.0f+10.0f*pow(sin(M_PI_CUDA*yiPlus1),2));
		}
	
		ynMinus1 = 1.0f + (x[n-1]+1.0f)/4.0f;
	
		f_x = (M_PI_CUDA/n)*(aux1 + aux2+pow(ynMinus1-1.0f,2));
	
		return f_x;
	}
};

double f_nelderMead(unsigned int n, const double *x, double *grad, void *f_data){
	return LevyMontalvo<double>()(x,n,f_data);
}

int main() {
	real T_0 = 1000, T_min = 0.01;
	const unsigned int n = 2, N = 100;
	const real rho = 0.99;
	size_t sizeFD = n * sizeof(real);
	real *lb, *ub, *cusimann_minimum = (real*)malloc(sizeFD), f_cusimann_minimum;
	lb = (real*)malloc(sizeFD);
	unsigned int i;
	for(i=0;i<n;i++)
		lb[i] = -10; 
	ub = (real*)malloc(sizeFD);
	for(i=0;i<n;i++)
		ub[i] = 10;

	unsigned int n_threads_per_block = 256;
	unsigned int n_blocks = 64;

	cusimann_optimize(n_threads_per_block, n_blocks, T_0, T_min, N, rho, n, lb, ub, LevyMontalvo<real>(), NULL, cusimann_minimum, &f_cusimann_minimum);

	printf("cusimann_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", cusimann_minimum[i]);
	printf(" ]\n");
	printf("f(cusimann_minimum) = %lf\n", f_cusimann_minimum);

	double *nelderMead_minimum = (double*)malloc(n*sizeof(double)), f_nelderMead_minimum;
	nelderMead_optimize(n, lb, ub, cusimann_minimum, f_nelderMead, NULL, nelderMead_minimum, &f_nelderMead_minimum);

	printf("nelderMead_minimum = [");
	for(i=0;i<n;i++)
		printf(" %f", nelderMead_minimum[i]);
	printf(" ]\n");
	printf("f(nelderMead_minimum) = %lf\n", f_nelderMead_minimum);

	free(lb);
	free(ub);
	free(cusimann_minimum);
	free(nelderMead_minimum);
	
	return EXIT_SUCCESS;
}
