#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo_reduction.cuh"

typedef float real;

// GPU outputs before CPU postprocessing
typedef struct
{
    real Expected;
    real Confidence;
} __TOptionValue;

const int THREAD_N = 256;

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut payoff functions for different precision modes
////////////////////////////////////////////////////////////////////////////////
__device__ inline float endCallValue(float S, float X, float r, float MuByT, float VBySqrtT)
{
    float callValue = S * __expf(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0F) ? callValue : 0.0F;
}

__device__ inline double endCallValue(double S, double X, double r, double MuByT, double VBySqrtT)
{
    double callValue = S * exp(MuByT + VBySqrtT * r) - X;
    return (callValue > 0.0) ? callValue     : 0.0;
}


static __global__ void rngSetupStates(
    unsigned int *d, 
    unsigned int *v, 
    int *boxmuller_flag, 
    int *boxmuller_flag_double, 
    float *boxmuller_extra, 
    double *boxmuller_extra_double, 
    int device_id)
{ 
    // determine global thread id 
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprandState rngState; 
    // TODO: check if this assignment is really necessary
    /*rngState.d = d[tid]; 
    for (int i = 0; i < 5; i++) rngState.v[i] = v[5*tid+i]; 
    rngState.boxmuller_flag = boxmuller_flag[tid]; 
    rngState.boxmuller_flag_double = boxmuller_flag_double[tid]; 
    rngState.boxmuller_extra = boxmuller_extra[tid]; 
    rngState.boxmuller_extra_double = boxmuller_extra_double[tid];
    */ 
    // Each threadblock gets different seed, 
    // Threads within a threadblock get different sequence numbers 
    hiprand_init(blockIdx.x + gridDim.x * device_id, threadIdx.x, 0, &rngState); 
    d[tid] = rngState.d; 
    for (int i = 0; i < 5; i++) v[5*tid+i] = rngState.v[i]; 
    boxmuller_flag[tid] = rngState.boxmuller_flag; 
    boxmuller_flag_double[tid] = rngState.boxmuller_flag_double; 
    boxmuller_extra[tid] = rngState.boxmuller_extra; 
    boxmuller_extra_double[tid] = rngState.boxmuller_extra_double; 
}


static __global__ void MonteCarloOneBlockPerOption( 
    unsigned int * __restrict d, 
    unsigned int * __restrict v, 
    int * __restrict boxmuller_flag, 
    int * __restrict boxmuller_flag_double,  
    float * __restrict boxmuller_extra, 
    double * __restrict boxmuller_extra_double, 
    float * optionData_S, 
    float * optionData_X, 
    float * optionData_MuByT, 
    float * optionData_VBySqrtT, 
    float * callValue_Expected, 
    float * callValue_Confidence, 
    int pathN, 
    int optionN) 
{ 
    const int SUM_N = THREAD_N; 
    __shared__ real s_SumCall[SUM_N]; 
    __shared__ real s_Sum2Call[SUM_N]; 
    // determine global thread id 
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    // reassemble hiprandState 
    hiprandState localState; 
    localState.d = d[tid]; 
    for (int i = 0; i < 5; i++) localState.v[i] = v[5*tid+i]; 
    localState.boxmuller_flag = boxmuller_flag[tid]; 
    localState.boxmuller_flag_double = boxmuller_flag_double[tid]; 
    localState.boxmuller_extra = boxmuller_extra[tid]; 
    localState.boxmuller_extra_double = boxmuller_extra_double[tid]; 
    for (int optionIndex = blockIdx.x; optionIndex < optionN; optionIndex += gridDim.x) 
    { 
        const real S = optionData_S[optionIndex]; 
        const real X = optionData_X[optionIndex]; 
        const real MuByT = optionData_MuByT[optionIndex]; 
        const real VBySqrtT = optionData_VBySqrtT[optionIndex]; 
        for (int iSum = threadIdx.x; iSum < SUM_N; iSum += blockDim.x) 
        { 
            __TOptionValue sumCall = {0, 0}; 
#pragma unroll 8 
            for (int i = iSum; i < pathN; i += SUM_N) 
            { 
                real r = hiprand_normal(&localState); 
                real callValue = endCallValue(S, X, r, MuByT, VBySqrtT); 
                sumCall.Expected += callValue; 
                sumCall.Confidence += callValue * callValue; 
            } 
            s_SumCall[iSum] = sumCall.Expected; 
            s_Sum2Call[iSum] = sumCall.Confidence; 
        } 
        // Reduce shared memory accumulators 
        // and write final result to global memory 
        sumReduce<real, SUM_N, THREAD_N>(s_SumCall, s_Sum2Call); 
        if (threadIdx.x == 0) 
        { 
            __TOptionValue t = {s_SumCall[0], s_Sum2Call[0]}; 
            callValue_Expected[optionIndex] = t.Expected; 
            callValue_Confidence[optionIndex] = t.Confidence; 
        } 
    } 
}

